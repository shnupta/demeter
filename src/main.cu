#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float T, r, sigma, dt, omega;


__device__ float normpdf(float x) {
  return exp(-0.5f * x * x) * (1.0f / sqrt(2.0f * M_PI));
}

////////////////////////////////////////////////////////////////////////
// kernel routine
////////////////////////////////////////////////////////////////////////


__global__ void pathcalc(float *d_z, float *d_v, float *d_delta, float *d_gamma)
{
  float s1, y1, payoff, avg_s1, delta, gamma;
  int   ind;

  // move array pointers to correct position

  // version 1
  ind = threadIdx.x + N*blockIdx.x*blockDim.x;

  // version 2
  // ind = 2*N*threadIdx.x + 2*N*blockIdx.x*blockDim.x;


  // path calculation

  s1 = 100.0f;
  avg_s1 = 100.0f;
  /* printf("Initial s1 = %f\n", s1); */

  for (int n=0; n<N; n++) {
    y1   = d_z[ind];
    // version 1
    ind += blockDim.x;      // shift pointer to next element

    s1 = s1 * (1.0f + r * dt + sigma * sqrt(dt) * y1);
    avg_s1 += s1;
    /* printf("New s1 = %f\n", s1); */
  }

  avg_s1 /= N;

  // put payoff value into device array

  /* payoff = avg_s1 - 100.0f > 0.0f ? exp(-r * T) : 0.0f; // binary asian */
  /* payoff = exp(-r * T) * max(s1 - 100.0f, 0.0f); */
  payoff = exp(-r * T) * max(avg_s1 - 100.0f, 0.0f); // arithmetic asian
  /* delta = s1 - 100.0f > 0.0f ? exp(-r * T) * (s1 / 100.0f) : 0.0f; */
  float psi_d = (log(100.0f) - log(avg_s1) - omega * dt) / (sigma * sqrt(dt));
  /* delta = (exp(-r * T) / 100.0f * sigma * sqrt(dt)) * normpdf(psi_d); // bin */
  delta = exp(r * (dt - T)) * (avg_s1 / 100.0f) * (1 - normcdf(psi_d - sigma * sqrt(dt))); // arith
  gamma = ((100.0f * exp(-r * T)) / (100.0f * 100.0f * sigma * sqrt(dt))) * normpdf(psi_d);
  /* printf("delta = %f\n", delta); */

  d_v[threadIdx.x + blockIdx.x*blockDim.x] = payoff;
  d_delta[threadIdx.x + blockIdx.x*blockDim.x] = delta;
  d_gamma[threadIdx.x + blockIdx.x*blockDim.x] = gamma;
}


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){
    
  int     NPATH=960000, h_N=100;
  /* int     NPATH=64, h_N=1; */
  float   h_T, h_r, h_sigma, h_dt, h_omega;
  float  *h_v, *d_v, *d_z, *h_delta, *d_delta, *h_gamma, *d_gamma;
  double  sum1, sum2;

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // allocate memory on host and device

  h_v = (float *)malloc(sizeof(float)*NPATH);
  h_delta = (float *)malloc(sizeof(float)*NPATH);
  h_gamma = (float *)malloc(sizeof(float)*NPATH);

  checkCudaErrors( hipMalloc((void **)&d_v, sizeof(float)*NPATH) );
  checkCudaErrors( hipMalloc((void **)&d_z, sizeof(float)*h_N*NPATH) );
  checkCudaErrors( hipMalloc((void **)&d_delta, sizeof(float)*h_N*NPATH) );
  checkCudaErrors( hipMalloc((void **)&d_gamma, sizeof(float)*h_N*NPATH) );

  // define constants and transfer to GPU

  h_T     = 1.0f;
  h_r     = 0.1f;
  h_sigma = 0.2f;
  h_dt    = h_T/h_N;
  h_omega = h_r - (h_sigma * h_sigma) / 2.0f;

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(T),    &h_T,    sizeof(h_T)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(r),    &h_r,    sizeof(h_r)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma),&h_sigma,sizeof(h_sigma)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dt),   &h_dt,   sizeof(h_dt)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(omega),   &h_omega,   sizeof(h_omega)) );

  // random number generation

  hipEventRecord(start);

  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
  checkCudaErrors( hiprandGenerateNormal(gen, d_z, h_N*NPATH, 0.0f, 1.0f) );
 
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
          milli, h_N*NPATH/(0.001*milli));

  // execute kernel and time it

  hipEventRecord(start);

  pathcalc<<<NPATH/64, 64>>>(d_z, d_v, d_delta, d_gamma);
  getLastCudaError("pathcalc execution failed\n");

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("Monte Carlo kernel execution time (ms): %f \n",milli);

  // copy back results

  checkCudaErrors( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
                   hipMemcpyDeviceToHost) );
  checkCudaErrors( hipMemcpy(h_delta, d_delta, sizeof(float)*NPATH,
                   hipMemcpyDeviceToHost) );
  checkCudaErrors( hipMemcpy(h_gamma, d_gamma, sizeof(float)*NPATH,
                   hipMemcpyDeviceToHost) );

  // compute average

  sum1 = 0.0;
  sum2 = 0.0;
  float deltasum = 0.0;
  float gammasum = 0.0;
  for (int i=0; i<NPATH; i++) {
    sum1 += h_v[i];
    sum2 += h_v[i]*h_v[i];
    deltasum += h_delta[i];
    gammasum += h_gamma[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
	 sum1/NPATH, sqrt((sum2/NPATH - (sum1/NPATH)*(sum1/NPATH))/NPATH) );
  printf("\nAverage delta = %13.8f\n\n", deltasum / NPATH);
  printf("\nAverage gamma = %13.8f\n\n", gammasum / NPATH);

  // Tidy up library

  checkCudaErrors( hiprandDestroyGenerator(gen) );

  // Release memory and exit cleanly

  free(h_v);
  free(h_delta);
  free(h_gamma);
  checkCudaErrors( hipFree(d_v) );
  checkCudaErrors( hipFree(d_z) );
  checkCudaErrors( hipFree(d_delta) );
  checkCudaErrors( hipFree(d_gamma) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

}

