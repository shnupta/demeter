#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>

#include "common.cuh"
#include "product.cuh"

////////////////////////////////////////////////////////////////////////
// kernel routine
////////////////////////////////////////////////////////////////////////

template <class T>
__global__ 
void pathcalc(float *d_z, mc_results<float> d_results) {
  T prod;
  // Index into random variables
  prod.ind = threadIdx.x + N*blockIdx.x*blockDim.x;

  prod.SimulatePath(N, d_z);
  prod.CalculatePayoffs(d_results);
}


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){
    
  int NPATH=960000, h_N=100;
  /* int     NPATH=64, h_N=1; */
  float h_T, h_r, h_sigma, h_dt, h_omega, h_s0, h_k;
  float *d_z;
  mc_results<float> h_results, d_results;
  double sum1, sum2, deltasum, vegasum, gammasum;

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // allocate memory on host and device

  h_results.AllocateHost(NPATH);
  d_results.AllocateDevice(NPATH);

  checkCudaErrors( hipMalloc((void **)&d_z, sizeof(float)*h_N*NPATH) );

  // define constants and transfer to GPU

  h_T     = 1.0f;
  h_r     = 0.1f;
  h_sigma = 0.2f;
  h_dt    = h_T/h_N;
  h_omega = h_r - (h_sigma * h_sigma) / 2.0f;
  h_s0      = 100.0f;
  h_k       = 100.0f;

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(T),    &h_T,    sizeof(h_T)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(r),    &h_r,    sizeof(h_r)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma),&h_sigma,sizeof(h_sigma)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dt),   &h_dt,   sizeof(h_dt)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(omega),   &h_omega,   sizeof(h_omega)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(s0),   &h_s0,   sizeof(h_s0)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(k),   &h_k,   sizeof(h_k)) );

  // random number generation

  hipEventRecord(start);

  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
  checkCudaErrors( hiprandGenerateNormal(gen, d_z, h_N*NPATH, 0.0f, 1.0f) );
 
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
          milli, h_N*NPATH/(0.001*milli));

  // execute kernel and time it

  printf("\n====== GPU ======\n");
  hipEventRecord(start);

  pathcalc< arithmetic_asian<float> > <<<NPATH/64, 64>>>(d_z, d_results);
  getLastCudaError("pathcalc execution failed\n");

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("Monte Carlo kernel execution time (ms): %f \n",milli);

  // copy back results

  h_results.CopyFromDevice(NPATH, d_results);

  // compute average

  sum1 = 0.0;
  sum2 = 0.0;
  deltasum = 0.0;
  gammasum = 0.0;
  for (int i=0; i<NPATH; i++) {
    sum1 += h_results.price[i];
    sum2 += h_results.price[i]*h_results.price[i];
    deltasum += h_results.delta[i];
    gammasum += h_results.gamma[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
	 sum1/NPATH, sqrt((sum2/NPATH - (sum1/NPATH)*(sum1/NPATH))/NPATH) );
  printf("Average delta = %13.8f\n\n", deltasum / NPATH);
  printf("Average gamma = %13.8f\n\n", gammasum / NPATH);

  // CPU calculation
  printf("====== CPU ======\n");

  // Copy random variables
  float *h_z = (float *) malloc(sizeof(float) * h_N * NPATH);
  checkCudaErrors( hipMemcpy(h_z, d_z, sizeof(float) * h_N * NPATH, hipMemcpyDeviceToHost) );

  arithmetic_asian<float> asian;
  auto h_start = std::chrono::steady_clock::now();
  asian.HostMC(NPATH, h_N, h_z, h_r, h_dt, h_sigma, h_s0, h_k, h_T, h_omega, h_results);
  auto h_end = std::chrono::steady_clock::now();
  float h_milli = std::chrono::duration_cast<std::chrono::milliseconds>(h_end - h_start).count();
  printf("CPU execution time (ms): %f \n", h_milli);


  sum1 = 0.0;
  sum2 = 0.0;
  deltasum = 0.0;
  gammasum = 0.0;
  for (int i=0; i<NPATH; i++) {
    sum1 += h_results.price[i];
    sum2 += h_results.price[i]*h_results.price[i];
    deltasum += h_results.delta[i];
    gammasum += h_results.gamma[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
	 sum1/NPATH, sqrt((sum2/NPATH - (sum1/NPATH)*(sum1/NPATH))/NPATH) );
  printf("Average delta = %13.8f\n\n", deltasum / NPATH);
  printf("Average gamma = %13.8f\n\n", gammasum / NPATH);


  printf("\nGPU speedup over serial CPU: %fx\n", h_milli / milli);

  // Tidy up library

  checkCudaErrors( hiprandDestroyGenerator(gen) );

  // Release memory and exit cleanly

  h_results.ReleaseHost();
  d_results.ReleaseDevice();
  free(h_z);

  checkCudaErrors( hipFree(d_z) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

}

